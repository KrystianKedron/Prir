#include "hip/hip_runtime.h"
#include<stdio.h>
#include"pomiar_czasu.h"
#include"pomiar_czasu.c"
#include "cuPrintf.cuh"
#include "cuPrintf.cu"

#define N 30
#define M 8 

// Sterowanie przedzialami
#define poczatek 0
#define koniec 1

double f1(double x) { 

	return 1/(1+x*x); 
}

__global__ void compute(float *calka, float *xp, float *h, int n) 
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	cuPrintf("Thread number %d. \n", index);
	cuPrintf("Xp = %f \n", xp);
//	if(index<n)
//		*calka = *calka + 1/(1+(*xp+index**h)*(*xp + index**h));
//		cuPrintf("Thread number %d. calka = %f\n", threadIdx.x, *calka);		
}

int main(void) {

	//PROGRAM SEKWENCYJNY (wersja na wzkaznikach)
	float val = 0;
	float *calka = &val;	

	float xk;
	
	float zero = poczatek;
	float *xp = &zero;

	xk = koniec;

	float steps = (xk - *xp) / (double)N;
	float *h = &steps;
	
	double czas_start = czas_zegara();
	
	for (int i=1; i<=N; i++)
	{
		*calka = *calka + f1(*xp + i**h);
	}

	*calka += (f1(*xp) + f1(xk)) / 2;
	*calka *= *h;
	*calka = 4**calka;
	
	printf("Calka sekwencyjna %f\n", *calka);

        double czas_stop = czas_zegara();
	double czas = czas_stop - czas_start;
        printf("Czas operacji na CPU %.4f \n", czas);

	// PROGRAM ROWNOLEGLY
	
	double czas_start_r = czas_zegara();
	int size = sizeof(float);
	float *d_xp, *d_h;
	float *d_calka, *r_calka;
	
	r_calka = (float *)malloc(size);
	xp = (float *)malloc(size);
	*xp = 10.00f;
	
	hipMalloc((void **)&d_calka, size);
	hipMalloc((void **)&d_xp, size);
	hipMalloc((void **)&d_h, size);	
	
	printf("Host xp = %f \n", *xp);	
	hipMemcpy(d_xp, xp, size, hipMemcpyHostToDevice);
	hipMemcpy(d_h, h, size, hipMemcpyHostToDevice);
	
	cudaPrintfInit();
	compute<<<(N+M-1)/M,M>>>(d_calka, d_xp, d_h, N);
	cudaPrintfDisplay(stdout, true);
    	cudaPrintfEnd();

	hipMemcpy(r_calka, d_calka, size, hipMemcpyDeviceToHost);
	printf("Calka przed rownolegla %f\n", *r_calka);

	//*r_calka += (f1(*xp) + f1(xk)) / 2;
	//*r_calka *= *h;
	//*r_calka = 4**r_calka;
	
	//printf("Calka rownolegla %f\n", *r_calka);
	double czas_stop_r = czas_zegara();
	double czas_r = czas_stop_r - czas_start_r;
        printf("Czas operacji na CUDA %.4f \n", czas_r);
	//free(xp); free(h); free(r_calka);
	hipFree(d_xp); hipFree(d_h); hipFree(d_calka);
	return 0;
}
