#include "hip/hip_runtime.h"
//Zrównoleglenie - kilka bloków - kilka wątków + równoważenie
#include<stdio.h>
#include"pomiar_czasu.h"
#include"pomiar_czasu.c"

#define N 300000000
#define M 8 //Watki na blok

__global__ void add(int *a, int *b, int *c, int n) 
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index<n)
		c[index] = a[index] + b[index];
}

void random (int *tab, int wym )
{	
	int i;
	for(i=0;i<wym;i++)
		tab[i]=rand()%101;
}

int main(void) {

	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	int i;
	int j;

	srand(time(NULL));
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random(a, N);
	b = (int *)malloc(size); random(b, N);
	c = (int *)malloc(size);
	
        int *d;
	
	double czas_start = czas_zegara();
        printf("Czas %.4f \n", czas_start);

        d = (int *)malloc(size);

        for (j = 0; j < N; j++){

                d[j] = a[j] + b[j];
        }

        for(i=0;i<N;i++)
        {
                //printf("a[%d](%d) + b[%d](%d) = d[%d](%d)\n",i,a[i],i,b[i],i,d[i]);
        }
	
        double czas_stop = czas_zegara();
	printf("Czas na CPU \n");
	printf("Czas stop %.4f \n", czas_stop);
	double czas = czas_stop - czas_start;
        printf("Czas %.4f \n", czas);

	i = 0;

	double czas_calosc_start = czas_zegara();
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	
	printf("Czas bez kopiowania na GPU \n");
	double czas_bez_start = czas_zegara();
        printf("Czas start %.4f \n", czas_bez_start);
	
	add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c, N);
	
	double czas_bez_stop = czas_zegara();
        printf("Czas stop %.4f \n", czas_bez_stop);
        double czas_bez = czas_bez_stop - czas_bez_start;
        printf("Czas %.4f \n", czas_bez);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	printf("Czas calosci operacji na GPU \n");
	printf("Czas start %.4f \n", czas_calosc_start);
	double czas_calosc_stop = czas_zegara();
	printf("Czas stop %.4f \n", czas_calosc_stop);
        double czas_calosc = czas_calosc_stop - czas_calosc_start;
        printf("Czas %.4f \n", czas_calosc);

	for(i=0;i<N;i++)
	{
		//printf("a[%d](%d) + b[%d](%d) = c[%d](%d)\n",i,a[i],i,b[i],i,c[i]);
	}
	// Cleanup
	//printf("%d+%d=%d\n",a,b,c);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	return 0;
}

